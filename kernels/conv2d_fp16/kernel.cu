#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "kernels.cu.h"
#include "gpu_utils.cu"

__global__ void conv2d_fp16_kernel(
    const __half* X, const __half* W, __half* Y,
    int N, int C, int H, int W_in,
    int K, int R, int S,
    int stride, int padding,
    int H_out, int W_out) {

    int y_w = blockIdx.x * blockDim.x + threadIdx.x;
    int y_h = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z % K;
    int n = blockIdx.z / K;

    if (y_h >= H_out || y_w >= W_out || n >= N) return;

    float sum = 0.0f;
    for (int c = 0; c < C; ++c) {
        for (int r = 0; r < R; ++r) {
            for (int s = 0; s < S; ++s) {
                int x_h = y_h * stride - padding + r;
                int x_w = y_w * stride - padding + s;
                if (x_h >= 0 && x_h < H && x_w >= 0 && x_w < W_in) {
                    int x_idx = n*(C*H*W_in) + c*(H*W_in) + x_h*W_in + x_w;
                    int w_idx = k*(C*R*S) + c*(R*S) + r*S + s;
                    sum += __half2float(X[x_idx]) * __half2float(W[w_idx]);
                }
            }
        }
    }
    int y_idx = n*(K*H_out*W_out) + k*(H_out*W_out) + y_h*W_out + y_w;
    Y[y_idx] = __float2half(sum);
}

void launch_conv2d_fp16(
    const uint16_t* h_X, const uint16_t* h_W, uint16_t* h_Y,
    int N, int C, int H, int W_in,
    int K, int R, int S,
    int stride, int padding) {

    const __half* X = reinterpret_cast<const __half*>(h_X);
    const __half* W = reinterpret_cast<const __half*>(h_W);
    __half* Y = reinterpret_cast<__half*>(h_Y);

    const int H_out = (H - R + 2 * padding) / stride + 1;
    const int W_out = (W_in - S + 2 * padding) / stride + 1;

    const size_t input_bytes = N * C * H * W_in * sizeof(__half);
    const size_t weight_bytes = K * C * R * S * sizeof(__half);
    const size_t output_bytes = N * K * H_out * W_out * sizeof(__half);

    __half *d_X, *d_W, *d_Y;
    CUDA_CHECK(hipMalloc(&d_X, input_bytes));
    CUDA_CHECK(hipMalloc(&d_W, weight_bytes));
    CUDA_CHECK(hipMalloc(&d_Y, output_bytes));

    CUDA_CHECK(hipMemcpy(d_X, X, input_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_W, W, weight_bytes, hipMemcpyHostToDevice));

    dim3 threads_per_block(16, 16, 1);
    dim3 blocks_in_grid(
        (W_out + threads_per_block.x - 1) / threads_per_block.x,
        (H_out + threads_per_block.y - 1) / threads_per_block.y,
        N * K
    );

    conv2d_fp16_kernel<<<blocks_in_grid, threads_per_block>>>(
        d_X, d_W, d_Y, N, C, H, W_in, K, R, S, stride, padding, H_out, W_out
    );
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(Y, d_Y, output_bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_Y));
}


