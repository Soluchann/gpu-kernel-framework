#include "hip/hip_runtime.h"
#include "eltwise.cu.h"
#include "eltwise_kernel.cu"
#include <iostream>

inline void CUDA_CHECK(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

void launch_eltwise_add_fp16(const __half* h_A, const __half* h_B, __half* h_Out, int N) {
    if (N % 2 != 0) {
        std::cerr << "Error: N must be even for half2 kernel" << std::endl;
        exit(1);
    }

    int N_half2 = N / 2;
    size_t bytes = N * sizeof(__half);

    __half *d_A = nullptr, *d_B = nullptr, *d_Out = nullptr;

    CUDA_CHECK(hipMalloc(&d_A, bytes));
    CUDA_CHECK(hipMalloc(&d_B, bytes));
    CUDA_CHECK(hipMalloc(&d_Out, bytes));

    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    int block_size = 512;  // higher occupancy than 256
    int grid_size = (N_half2 + block_size - 1) / block_size;

    // Setup CUDA events for timing kernel only
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    hipEventRecord(start);
    eltwise_add_kernel_half2<<<grid_size, block_size>>>(
        reinterpret_cast<const half2*>(d_A),
        reinterpret_cast<const half2*>(d_B),
        reinterpret_cast<half2*>(d_Out),
        N_half2);
    CUDA_CHECK(hipGetLastError());
    hipEventRecord(stop);

    CUDA_CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));

    // Copy result back
    CUDA_CHECK(hipMemcpy(h_Out, d_Out, bytes, hipMemcpyDeviceToHost));

    // Cleanup
    hipFree(d_A); 
    hipFree(d_B); 
    hipFree(d_Out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}