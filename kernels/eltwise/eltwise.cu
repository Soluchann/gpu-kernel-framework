#include "hip/hip_runtime.h"
#include "eltwise.cu.h"
#include "eltwise_kernel.cu"
#include <iostream>

void CUDA_CHECK(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

void launch_eltwise_add_fp16(const __half* h_A, const __half* h_B, __half* h_Out, int N) {
    __half *d_A, *d_B, *d_Out;
    size_t bytes = N * sizeof(__half);

    CUDA_CHECK(hipMalloc(&d_A, bytes));
    CUDA_CHECK(hipMalloc(&d_B, bytes));
    CUDA_CHECK(hipMalloc(&d_Out, bytes));

    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;
    eltwise_add_kernel<<<grid_size, block_size>>>(d_A, d_B, d_Out, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_Out, d_Out, bytes, hipMemcpyDeviceToHost));

    hipFree(d_A); hipFree(d_B); hipFree(d_Out);
}
