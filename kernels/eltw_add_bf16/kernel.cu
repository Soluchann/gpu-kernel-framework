#include "hip/hip_runtime.h"
// kernels/eltwise/eltwise_bf16.cu
#include "kernels.cu.h"
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include "gpu_utils.cu"

__global__ void eltwise_add_bf16_kernel(
    const __hip_bfloat16* A,
    const __hip_bfloat16* B,
    __hip_bfloat16* Out,
    int N
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        Out[idx] = __hadd(A[idx], B[idx]);
    }
}

void launch_eltwise_add_bf16(
    const uint16_t* h_A,
    const uint16_t* h_B,
    uint16_t* h_Out,
    int N
) {
    const __hip_bfloat16* A = reinterpret_cast<const __hip_bfloat16*>(h_A);
    const __hip_bfloat16* B = reinterpret_cast<const __hip_bfloat16*>(h_B);
    __hip_bfloat16* Out = reinterpret_cast<__hip_bfloat16*>(h_Out);

    size_t bytes = N * sizeof(__hip_bfloat16);
    __hip_bfloat16 *d_A, *d_B, *d_Out;

    CUDA_CHECK(hipMalloc(&d_A, bytes));
    CUDA_CHECK(hipMalloc(&d_B, bytes));
    CUDA_CHECK(hipMalloc(&d_Out, bytes));

    CUDA_CHECK(hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice));

    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;
    eltwise_add_bf16_kernel<<<grid_size, block_size>>>(d_A, d_B, d_Out, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(Out, d_Out, bytes, hipMemcpyDeviceToHost));

    hipFree(d_A); hipFree(d_B); hipFree(d_Out);
}